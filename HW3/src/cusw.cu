#include "hip/hip_runtime.h"
// optimized_cuda_sw.cu

#include "sw.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

struct MaxScore {
    int score;
    int i;
    int j;
};

__device__ int dev_max4(int a, int b, int c, int d) {
    return max(max(a, b), max(c, d));
}

__global__ void cuda_sw_kernel_expand(
    int* H, int* E, int* F, const char* ref, const char* query, int k,
    int M, int N, int match, int mismatch, int gap_open, int gap_extend, MaxScore* d_max
) {
    // i: ref, j:query, k:thread or 斜對角 idx (start on 1)
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int i = tid + 1;
    int j = k - i + 1;

    if (i > 0 && j > 0 && i <= M && j <= N) {
        int idx = j * (M+1) + i;
        int idx_up = (j-1) * (M+1) + i;
        int idx_left = j * (M+1) + (i-1);
        int idx_diag = (j-1) * (M+1) + (i-1);

        int sub_score = (ref[i-1] == query[j-1]) ? match : -mismatch;
        int e = max(H[idx_left] - gap_open, E[idx_left] - gap_extend);
        int f = max(H[idx_up] - gap_open, F[idx_up] - gap_extend);
        int h = dev_max4(0, H[idx_diag] + sub_score, e, f);

        H[idx] = h;
        E[idx] = e;
        F[idx] = f;

        // 儲存最大值（只在值變大時記錄座標）
        int old = atomicMax(&(d_max->score), h);
        if (h > old) {
            d_max->i = i;
            d_max->j = j;
        }
    }
}

__global__ void cuda_sw_kernel_shrink(
    int* H, int* E, int* F, const char* ref, const char* query, int k,
    int M, int N, int match, int mismatch, int gap_open, int gap_extend, MaxScore* d_max
) {
    int i_start = max(1, k - N + 1);
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int i = tid + i_start;
    int j = k + 1 - i;
    //printf("[k=%d] thread %d → i=%d, j=%d\n", k, threadIdx.x, i, j);
    
    if (i > 0 && j > 0 && i <= M && j <= N) {
        int idx = j * (M+1) + i;
        int idx_up = (j-1) * (M+1) + i;
        int idx_left = j * (M+1) + (i-1);
        int idx_diag = (j-1) * (M+1) + (i-1);

        int sub_score = (ref[i-1] == query[j-1]) ? match : -mismatch;
        int e = max(H[idx_left] - gap_open, E[idx_left] - gap_extend);
        int f = max(H[idx_up] - gap_open, F[idx_up] - gap_extend);
        int h = dev_max4(0, H[idx_diag] + sub_score, e, f);

        H[idx] = h;
        E[idx] = e;
        F[idx] = f;

        // 儲存最大值（只在值變大時記錄座標）
        int old = atomicMax(&(d_max->score), h);
        if (h > old) {
            d_max->i = i;
            d_max->j = j;
        }
    }
}


SmithWaterman cuda_smith_waterman(std::string_view ref, std::string_view query,
    int match, int mismatch, int gap_open, int gap_extend
) {
    int M = ref.size();
    int N = query.size();

    std::vector<int> H((M+1)*(N+1), 0);
    std::vector<int> E((M+1)*(N+1), 0);
    std::vector<int> F((M+1)*(N+1), 0);

    int max_score = 0;
    int max_i = 0, max_j = 0;

    int *dpH_dev, *dpE_dev, *dpF_dev;
    char *dev_ref, *dev_query;
    MaxScore* d_max;
    MaxScore h_max = {0, 0, 0};


    hipMalloc(&dpH_dev, (M+1)*(N+1)*sizeof(int));
    hipMalloc(&dpE_dev, (M+1)*(N+1)*sizeof(int));
    hipMalloc(&dpF_dev, (M+1)*(N+1)*sizeof(int));
    hipMalloc(&dev_ref, M * sizeof(char));
    hipMalloc(&dev_query, N * sizeof(char));
    hipMalloc(&d_max, sizeof(MaxScore));

    hipMemcpy(dpH_dev, H.data(), (M+1)*(N+1)*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dpE_dev, E.data(), (M+1)*(N+1)*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dpF_dev, F.data(), (M+1)*(N+1)*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_ref, ref.data(), M, hipMemcpyHostToDevice);
    hipMemcpy(dev_query, query.data(), N, hipMemcpyHostToDevice);
    hipMemcpy(d_max, &h_max, sizeof(MaxScore), hipMemcpyHostToDevice);

	// kernel implement
    int threadsPerBlock = 1024;
    for (int k = 1; k <= min(M, N); ++k) {
        int totalThreads = k;
        int numBlocks = (totalThreads + threadsPerBlock - 1) / threadsPerBlock;
    
        cuda_sw_kernel_expand<<<numBlocks, threadsPerBlock>>>(
            dpH_dev, dpE_dev, dpF_dev, dev_ref, dev_query, 
            k, M, N, match, mismatch, gap_open, gap_extend, d_max
        );
        hipDeviceSynchronize();
    }
    
    for (int k = min(M, N) + 1; k <= M + N - 1; ++k) {
        int i_start = max(1, k - N + 1);
        int i_end = min(M, k);
        int thread_count = i_end - i_start + 1;
        int numBlocks = (thread_count + threadsPerBlock - 1) / threadsPerBlock;
    
        cuda_sw_kernel_shrink<<<numBlocks, threadsPerBlock>>>(
            dpH_dev, dpE_dev, dpF_dev, dev_ref, dev_query,
            k, M, N, match, mismatch, gap_open, gap_extend, d_max
        );
        hipDeviceSynchronize();
    }    

    hipMemcpy(H.data(), dpH_dev, (M+1)*(N+1)*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(E.data(), dpE_dev, (M+1)*(N+1)*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(F.data(), dpF_dev, (M+1)*(N+1)*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&h_max, d_max, sizeof(MaxScore), hipMemcpyDeviceToHost);

    hipFree(dpH_dev);
    hipFree(dpE_dev);
    hipFree(dpF_dev);
    hipFree(dev_ref);
    hipFree(dev_query);
    hipFree(d_max);

    // Print H matrix
    //printf("\nScoring matrix H:\n");
    //for (int i = 0; i <= M; ++i) {
    //    for (int j = 0; j <= N; ++j) {
    //        printf("%3d ", H[j*(M+1)+i]);
    //    }
    //    printf("\n");
    //}

    std::string aligned_ref, aligned_query, match_line;
    int i = max_i = h_max.i, j = max_j = h_max.j;
    max_score = h_max.score;
    while (i > 0 && j > 0 && H[j * (M+1) + i] > 0) {
        int idx      = j * (M+1) + i;
        int idx_diag = (j-1) * (M+1) + (i-1);
        int idx_up   = (j-1) * (M+1) + i;
        int idx_left = j * (M+1) + (i-1);
    
        int score_diag = H[idx_diag] + ((ref[i-1] == query[j-1]) ? match : -mismatch);
        int score_E = std::max(H[idx_left] - gap_open, E[idx_left] - gap_extend);
        int score_F = std::max(H[idx_up] - gap_open, F[idx_up] - gap_extend);
    
        if (H[idx] == score_E) {
            aligned_ref += ref[i-1];
            aligned_query += '-';
            match_line += ' ';
            i--;
        } else if (H[idx] == score_F) {
            aligned_ref += '-';
            aligned_query += query[j-1];
            match_line += ' ';
            j--;
        } else {
            aligned_ref += ref[i-1];
            aligned_query += query[j-1];
            match_line += (ref[i-1] == query[j-1] ? '|' : '*');
            i--; j--;
        }
    }    

    std::reverse(aligned_ref.begin(), aligned_ref.end());
    std::reverse(aligned_query.begin(), aligned_query.end());
    std::reverse(match_line.begin(), match_line.end());

    return SmithWaterman{
        .score = static_cast<int16_t>(max_score),
        .aligned_seq1 = std::move(aligned_ref),
        .aligned_seq2 = std::move(aligned_query),
        .match_line = std::move(match_line),
        .start1 = static_cast<size_t>(i),
        .end1 = static_cast<size_t>(max_i),
        .start2 = static_cast<size_t>(j),
        .end2 = static_cast<size_t>(max_j)
    };
}