#include "hip/hip_runtime.h"
#include "sw.hpp"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#define DIR_NONE  0  // 來自 0，起點
#define DIR_DIAG  1  // 來自斜對角（配對/不配）
#define DIR_LEFT  2  // 來自左（gap in query）
#define DIR_UP    3  // 來自上（gap in ref）

extern "C" void cuda_warmup() {
    hipFree(0); // 觸發初始化
}

struct MaxScore {
    int score;
    int i;
    int j;
};

__device__ void update_max_score(MaxScore* d_max, int h, int i, int j) {
    int old = atomicMax(&(d_max->score), h);
    if (h > old) {
        d_max->i = i;
        d_max->j = j;
    }
}

__device__ int dev_max4(int a, int b, int c, int d) {
    return max(max(a, b), max(c, d));
}

// kernel to compute diagonal wavefront, only using 3 row buffers
__global__ void cuda_sw_kernel(
    const char* ref, const char* query,
    int* H_prev2, int* H_prev1, int* H_curr,
    int* E_prev, int* E_curr,
    int* F_prev, int* F_curr,
    uint8_t* d_dir,
    int k, int M, int N,
    int match, int mismatch,
    int gap_open, int gap_extend,
    MaxScore* d_max
) {
    // i + j = k + 1
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int i_start = max(1, k - N + 1);  // i 起點
    int i_end   = min(M, k);          // i 終點
    int i = i_start + tid;
    int j = k - i + 1;

    if (i <=0 || i > i_end || j <= 0 || j > N) return; 

    int idx_up = i;
    int idx_left = i - 1;
    int idx_diag = i - 1;

    int sub_score = (ref[i-1] == query[j-1]) ? match : -mismatch;
    int e = max(H_prev1[idx_left] - gap_open, E_prev[idx_left] - gap_extend);
    int f = max(H_prev1[idx_up] - gap_open, F_prev[idx_up] - gap_extend);
    int h = dev_max4(0, H_prev2[idx_diag] + sub_score, e, f);

    H_curr[i] = h;
    E_curr[i] = e;
    F_curr[i] = f;

    // 決定方向
    uint8_t dir = DIR_NONE;
    int h_diag = H_prev2[idx_diag] + sub_score;
    if (h == e) dir = DIR_LEFT;
    else if (h == f) dir = DIR_UP;
    else if (h == h_diag) dir = DIR_DIAG;
    d_dir[j * (M + 1) + i] = dir;  // 對應於 direction[j][i]

    update_max_score(d_max, h, i, j);
}

SmithWaterman cuda_smith_waterman(std::string_view ref, std::string_view query,
    int match, int mismatch, int gap_open, int gap_extend
) {
    int M = ref.size();
    int N = query.size();
    int size = max(M, N);

    std::vector<int> H_prev2(size  + 1, 0); std::vector<int> H_prev1(size  + 1, 0); std::vector<int> H_curr(size  + 1, 0);
    std::vector<int> E_prev(size  + 1, 0); std::vector<int> E_curr(size  + 1, 0);
    std::vector<int> F_prev(size  + 1, 0); std::vector<int> F_curr(size  + 1, 0);
    std::vector<uint8_t> flat_dir((M + 1) * (N + 1));
    
    int *d_H_prev2; int *d_H_prev1; int *d_H_curr;
    int *d_E_prev; int *d_E_curr;
    int *d_F_prev; int *d_F_curr;    
    char *d_ref, *d_query;
    MaxScore* d_max;
    MaxScore h_max = {0, 0, 0};
    uint8_t* d_dir;
    
    // malloc
    hipMalloc(&d_H_prev2, sizeof(int) * (size  + 1)); hipMalloc(&d_H_prev1, sizeof(int) * (size  + 1)); hipMalloc(&d_H_curr,  sizeof(int) * (size  + 1));
    hipMalloc(&d_E_prev,  sizeof(int) * (size  + 1)); hipMalloc(&d_E_curr,  sizeof(int) * (size  + 1));
    hipMalloc(&d_F_prev,  sizeof(int) * (size  + 1)); hipMalloc(&d_F_curr,  sizeof(int) * (size  + 1));
    hipMalloc(&d_ref, sizeof(char) * M); hipMalloc(&d_query, sizeof(char) * N); hipMalloc(&d_max, sizeof(MaxScore));
    hipMalloc(&d_dir, sizeof(uint8_t) * (M + 1) * (N + 1));

    // memcpy
    hipMemcpy(d_H_prev2, H_prev2.data(), sizeof(int) * (size  + 1), hipMemcpyHostToDevice); hipMemcpy(d_H_prev1, H_prev1.data(), sizeof(int) * (size  + 1), hipMemcpyHostToDevice); hipMemcpy(d_H_curr, H_curr.data(), sizeof(int) * (size  + 1), hipMemcpyHostToDevice);
    hipMemcpy(d_E_prev, E_prev.data(), sizeof(int) * (size  + 1), hipMemcpyHostToDevice); hipMemcpy(d_E_curr, E_curr.data(), sizeof(int) * (size  + 1), hipMemcpyHostToDevice);
    hipMemcpy(d_F_prev, F_prev.data(), sizeof(int) * (size  + 1), hipMemcpyHostToDevice); hipMemcpy(d_F_curr, F_curr.data(), sizeof(int) * (size  + 1), hipMemcpyHostToDevice);
    hipMemcpy(d_ref, ref.data(), M, hipMemcpyHostToDevice); hipMemcpy(d_query, query.data(), N, hipMemcpyHostToDevice); 
    hipMemcpy(d_max, &h_max, sizeof(MaxScore), hipMemcpyHostToDevice);

    // kernel implement
    for (int k = 1; k< M + N - 1; ++k) {
        int i_start = max(1, k - N + 1);
        int i_end = min(M, k);
        int thread_count = i_end - i_start + 1;
    
        int threadsPerBlock = 256;
        int numBlocks = (thread_count + threadsPerBlock - 1) / threadsPerBlock;
    
        cuda_sw_kernel<<<numBlocks, threadsPerBlock>>>(
            d_ref, d_query,
            d_H_prev2, d_H_prev1, d_H_curr,
            d_E_prev, d_E_curr,
            d_F_prev, d_F_curr,
            d_dir,
            k, M, N, match, mismatch, gap_open, gap_extend,
            d_max
        );

        std::swap(d_H_prev2, d_H_prev1);
        std::swap(d_H_prev1, d_H_curr);
        std::swap(d_E_prev, d_E_curr);
        std::swap(d_F_prev, d_F_curr);
    }
    hipDeviceSynchronize();

    hipMemcpy(&h_max, d_max, sizeof(MaxScore), hipMemcpyDeviceToHost);
    hipMemcpy(flat_dir.data(), d_dir, sizeof(uint8_t) * (M + 1) * (N + 1), hipMemcpyDeviceToHost);
    
    // clean up
    hipFree(d_H_prev2); hipFree(d_H_prev1); hipFree(d_H_curr);
    hipFree(d_E_prev); hipFree(d_E_curr);
    hipFree(d_F_prev); hipFree(d_F_curr);
    hipFree(d_ref); hipFree(d_query);
    hipFree(d_max);
    hipFree(d_dir);


    //direction[j][i] = flat_dir[j * (M + 1) + i];
    std::string aligned_ref, aligned_query, match_line;
    int i = h_max.i, j = h_max.j;
    while (i > 0 && j > 0) {
        uint8_t dir = flat_dir[j * (M + 1) + i];
        if (dir == DIR_LEFT) {
            aligned_ref += ref[i - 1];
            aligned_query += '-';
            match_line += ' ';
            --i;
        } else if (dir == DIR_UP) {
            aligned_ref += '-';
            aligned_query += query[j - 1];
            match_line += ' ';
            --j;
        } else if (dir == DIR_DIAG) {
            aligned_ref += ref[i - 1];
            aligned_query += query[j - 1];
            match_line += (ref[i - 1] == query[j - 1] ? '|' : '*');
            --i; --j;
        } else break;  // DIR_NONE or invalid
    }
    std::reverse(aligned_ref.begin(), aligned_ref.end());
    std::reverse(aligned_query.begin(), aligned_query.end());
    std::reverse(match_line.begin(), match_line.end());
    
    return SmithWaterman{
        .score = static_cast<int16_t>(h_max.score),
        .aligned_seq1 = std::move(aligned_ref),
        .aligned_seq2 = std::move(aligned_query),
        .match_line = std::move(match_line),
        .start1 = static_cast<size_t>(i),
        .end1 = static_cast<size_t>(h_max.i),
        .start2 = static_cast<size_t>(j),
        .end2 = static_cast<size_t>(h_max.j)
    };

    //return SmithWaterman{
    //    .score = h_max.score,
    //    .aligned_seq1 = "",
    //    .aligned_seq2 = "",
    //    .match_line = "",
    //    .start1 = 0,
    //    .end1 = static_cast<size_t> (h_max.i),
    //    .start2 = 0,
    //    .end2 = static_cast<size_t> (h_max.j)
    //};
}